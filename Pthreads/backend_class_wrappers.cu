///
/// \author Poutas Sokratis (sokratispoutas@gmail.com)
///
/// \brief 
///

#include <queue>
#include <unihelpers.hpp>
#include <sstream>
#include <backend_wrappers.hpp>
#include "queues_per_device.hpp"

int lvl = 1;

int Event_num_device[128] = {0};
#ifndef UNIHELPER_LOCKFREE_ENABLE
int unihelper_lock = 0;
#endif

inline void get_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	while(__sync_lock_test_and_set (&unihelper_lock, 1)){
		;
		#ifdef UDDEBUG
			lprintf(lvl, "------- Spinning on Unihelper lock\n");
		#endif
	}
#endif
	;
}
inline void release_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	__sync_lock_release(&unihelper_lock);
#endif
	;
}

/*****************************************************/
/// Event Status-related functions

const char* print_event_status(event_status in_status){
	switch(in_status){
		case(UNRECORDED):
			return "UNRECORDED";
		case(RECORDED):
			return "RECORDED";
		case(COMPLETE):
			return "COMPLETE";
		case(CHECKED):
			return "CHECKED";
		case(GHOST):
			return "GHOST";
		default:
			error("print_event_status: Unknown state\n");
	}
}

void* taskExecLoop(void * args)
{
	// extract queue and lock from data
	queue_data_p thread_data = (queue_data_p) args;
	std::queue<pthread_task_p>* task_queue_p = (std::queue<pthread_task_p>* )thread_data->taskQueue;

	while(1){
		get_lock_q(&thread_data->queueLock);
		if(thread_data->terminate){
			release_lock_q(&thread_data->queueLock);
			break;
		} 
		else if(task_queue_p->size() > 0){
			for(int i = 0; i < STREAM_POOL_SZ; i++)
				massert(hipSuccess == hipStreamQuery(thread_data->stream_pool[i]), "Error: Found stream with pending work\n");

			// get next task
			pthread_task_p curr_task_p = task_queue_p->front();
			release_lock_q(&thread_data->queueLock);

			if(curr_task_p){
				#ifdef UDDEBUG
					std::stringstream inMsg;
					inMsg << "|-----> taskExecLoop(thread = " << thread_data->threadId << "): function = " << curr_task_p->function_name << "\n";
					std::cout << inMsg.str();
				#endif
				// execute task
				void* (*curr_func) (void*);
				curr_func = (void* (*)(void*))curr_task_p->func;
				curr_func(curr_task_p->data);
				#ifdef UDDEBUG
					std::stringstream outMsg;
					outMsg << "<-----| taskExecLoop(thread = " << thread_data->threadId << "): function = " << curr_task_p->function_name << "\n";
					std::cout << outMsg.str();
				#endif

				for(int i = 0; i < STREAM_POOL_SZ; i++)
					massert(hipSuccess == hipStreamSynchronize(thread_data->stream_pool[i]), "Error: while synchronizing stream %d\n", i);

				get_lock_q(&thread_data->queueLock);
				if(task_queue_p->size() > 0)
					task_queue_p->pop();
				else{
					std::stringstream errorMsg;
					errorMsg << "taskExecLoop: Error: Thread " << thread_data->threadId << " -- tried to pop from empty queue" << "\n";
					std::cout << errorMsg.str();
				}
				release_lock_q(&thread_data->queueLock);

				// delete task
				delete(curr_task_p);
			}
			else{
				// This should not happen
				std::stringstream errorMsg;
				errorMsg << "taskExecLoop: Error: Thread " << thread_data->threadId << " -- task = " << curr_task_p << "\n" << "taskExecLoop: Shouldn't reach this point " << "\n";
				std::cout << errorMsg.str();
			}
		}
		else{
			release_lock_q(&thread_data->queueLock);
		}
	}

	return 0;
}

/*****************************************************/
/// Command queue class functions
CommandQueue::CommandQueue(int dev_id_in)
{
	int prev_dev_id = CoCoPeLiaGetDevice();
	dev_id = dev_id_in;
	CoCoPeLiaSelectDevice(dev_id);
	if(!queuesPerDeviceInitialized){
		InitializeQueuesPerDevice();
	}
	AssignQueueToDevice(this, dev_id);
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::CommandQueue()\n", dev_id_in);
#endif
	if(prev_dev_id != dev_id){;
#ifdef UDEBUG
		lprintf(lvl, "[dev_id=%3d] ------- CommandQueue::CommandQueue(): Called for other dev_id = %d\n",
			dev_id, prev_dev_id);
#endif
	}
	
#ifdef UDEBUG
		lprintf(lvl, "[dev_id=%3d] ------- CommandQueue::CommandQueue(%d): Initializing simple queue\n", dev_id);
#endif
	// Create stream pool
	hipStream_t* stream_pool = (hipStream_t*)malloc(STREAM_POOL_SZ * sizeof(hipStream_t));
	for(int i = 0; i < STREAM_POOL_SZ; i++){
		hipError_t err = hipStreamCreate(&stream_pool[i]);
		massert(hipSuccess == err, "CommandQueue::CommandQueue(%d) - %s\n", dev_id, hipGetErrorString(err));
	}

	// Create cublas handle
	hipblasHandle_t* handle_p = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
	massert(HIPBLAS_STATUS_SUCCESS == hipblasCreate(handle_p),
		"CommandQueue::CommandQueue(%d): hipblasCreate failed\n", dev_id);


	std::queue<pthread_task_p>* task_queue = new std::queue<pthread_task_p>;
	cqueue_backend_ptr = (void *) task_queue;
	queue_data_p data = new queue_data;
	
	data->taskQueue = (void *) task_queue;
	data->queueLock = 0; // initialize queue lock
	data->terminate = false;
	data->stream_pool = stream_pool;
	data->stream_ctr = 0;
	data->handle_p = handle_p;
	cqueue_backend_data = (void*) data;

	// Spawn thread that loops over queue and executes tasks
	if(pthread_create(&(data->threadId), NULL, taskExecLoop, data)) std::cout << "Error: CommandQueue::CommandQueue: pthread_create failed" << std::endl;

	// std::cout << "CommandQueue::CommandQueue: Queue constructor complete. Thread id = " << data->threadId << std::endl;

	CoCoPeLiaSelectDevice(prev_dev_id);
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::CommandQueue()\n", dev_id);
#endif
}

CommandQueue::~CommandQueue()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::~CommandQueue()\n", dev_id);
#endif
	sync_barrier();
	CoCoPeLiaSelectDevice(dev_id);
	UnassignQueueFromDevice(this, dev_id);

	queue_data_p backend_d = (queue_data_p) cqueue_backend_data;
	for(int i = 0; i < STREAM_POOL_SZ; i++){
		massert(hipSuccess == hipStreamQuery(backend_d->stream_pool[i]), "CommandQueue::~CommandQueue: Found stream with pending work\n");
	}

	get_lock_q(&backend_d->queueLock);
	backend_d->terminate = true;
	release_lock_q(&backend_d->queueLock);

	if(pthread_join(backend_d->threadId, NULL)) std::cout << "Error: CommandQueue::~CommandQueue: pthread_join failed" << std::endl;

	std::queue<pthread_task_p> * task_queue_p = (std::queue<pthread_task_p> *)cqueue_backend_ptr;

	for(int i = 0; i < STREAM_POOL_SZ; i++){
		massert(hipSuccess == hipStreamQuery(backend_d->stream_pool[i]), "About to destroy stream with pending work\n");
		hipError_t err = hipStreamDestroy(backend_d->stream_pool[i]);
		massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamDestroy: %s\n", hipGetErrorString(err));
	}
	massert(HIPBLAS_STATUS_SUCCESS == hipblasDestroy(*(backend_d->handle_p)),
		"CommandQueue::~CommandQueue - hipblasDestroy(handle) failed\n");

	delete(task_queue_p);
	delete(backend_d);

#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::~CommandQueue()\n", dev_id);
#endif
	return;
}

void CommandQueue::sync_barrier()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::sync_barrier()\n", dev_id);
#endif

	std::queue<pthread_task_p> * task_queue_p = (std::queue<pthread_task_p> *)cqueue_backend_ptr;
	queue_data_p backend_d = (queue_data_p) cqueue_backend_data;

	bool queueIsBusy = true;
	// busy wait until task queue is empty
	while(queueIsBusy){
		get_lock_q(&backend_d->queueLock);
		queueIsBusy = task_queue_p->size() > 0;
		if(!queueIsBusy){
			for(int i = 0; i < STREAM_POOL_SZ; i++)
				massert(hipSuccess == hipStreamSynchronize(backend_d->stream_pool[i]), "Error: while synchronizing stream %d\n", i);
		}
		release_lock_q(&backend_d->queueLock);
	}

	// std::cout << "CommandQueue::sync_barrier: sync_barrier complete" << std::endl;

#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::sync_barrier()\n", dev_id);
#endif
}

void CommandQueue::add_host_func(void* func, void* data, std::string name, std::string caller){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::add_host_func() getting lock\n", dev_id);
#endif
	get_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] ------- CommandQueue::add_host_func()\n", dev_id);
#endif

	std::queue<pthread_task_p> * task_queue_p = (std::queue<pthread_task_p> *)cqueue_backend_ptr;
	pthread_task_p task_p = new pthread_task;
	task_p->func = func;
	task_p->data = data;
	task_p->function_name = name;
	// if(name.compare("Default_name") == 0) std::cout << "CommandQueue::add_host_func() called with default function name from caller = " << caller << "\n";

	queue_data_p backend_d = (queue_data_p) cqueue_backend_data;


	get_lock_q(&backend_d->queueLock);
	task_queue_p->push(task_p);
	release_lock_q(&backend_d->queueLock);

	release_lock();

#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::add_host_func()\n", dev_id);
#endif
}

void * blockQueue(void * data){
	Event_p Wevent = (Event_p) data;

	while(Wevent->query_status() < COMPLETE){
		;
	}

	#ifdef DDEBUG
		lprintf(lvl, "[dev_id=%3d] <-----| blockQueue(Event(%d)): done blocking for event = %p\n", Wevent->dev_id, Wevent->id, Wevent);
	#endif
	return 0;
}

void CommandQueue::wait_for_event(Event_p Wevent)
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::wait_for_event(Event(%d))\n", dev_id, Wevent->id);
#endif
	if (Wevent->query_status() == CHECKED);
	else{
		// TODO: New addition (?)
		if (Wevent->query_status() == UNRECORDED) {
			warning("CommandQueue::wait_for_event():: UNRECORDED event\n");
			return;
		}

		#ifdef DDEBUG
			lprintf(lvl, "CommandQueue::wait_for_event event = %p (status = %s) : queue = %p\n", Wevent, print_event_status(Wevent->query_status()), this);
		#endif
		add_host_func((void*) &blockQueue, (void*) Wevent, "blockQueue");
	}
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::wait_for_event(Event(%d))\n", dev_id, Wevent->id);
#endif
	return;
}

void* eventFunc(void* event_data){
	pthread_event_p event_p = (pthread_event_p) event_data;
	event_p->estate = COMPLETE;
	event_p->completeTime = std::chrono::steady_clock::now();

	return 0;
}


/*****************************************************/
/// Event class functions. TODO: Do status = .. commands need lock?
Event::Event(int dev_id_in)
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::Event()\n", dev_id_in, Event_num_device[idxize(dev_id_in)]);
#endif
	get_lock();
	id = Event_num_device[idxize(dev_id_in)];
	Event_num_device[idxize(dev_id_in)]++;
	dev_id = dev_id_in - 42;

	pthread_event_p event_p = new pthread_event;
	event_p->estate = UNRECORDED;
	event_backend_ptr = (void*) event_p;
	status = UNRECORDED;
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::Event()\n", dev_id, id);
#endif
}

Event::~Event()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::~Event()\n", dev_id, id);
#endif
	sync_barrier();
	// std::cout << "Event::~Event: waiting for unihelpersLock" << std::endl;
	get_lock();
	if (dev_id < -1) 	Event_num_device[idxize(dev_id+42)]--;
	else Event_num_device[idxize(dev_id)]--;

	pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
	delete(event_p);
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::~Event()\n", dev_id, id);
#endif
}

void Event::sync_barrier()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::sync_barrier()\n", dev_id, id);
#endif
	//get_lock();
	if (status != CHECKED){
		if (status == UNRECORDED){;
#ifdef UDEBUG
			warning("[dev_id=%3d] |-----> Event(%d)::sync_barrier() - Tried to sync unrecorded event\n", dev_id, id);
#endif
		}
		else{
			pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
			#ifdef DEBUG
				lprintf(lvl, "|-----> Event(%p)::sync_barrier() started waiting... state = %s\n", this, print_event_status(event_p->estate));
			#endif
			while(query_status() < COMPLETE){;
				#ifdef UDDEBUG
					lprintf(lvl, "[dev_id=%3d] ------- Event(%d)::sync_barrier() waiting... state = %s\n", dev_id, id, print_event_status(event_p->estate));
				#endif
			}

			if (status == RECORDED){ 
				status = CHECKED;
				event_p->estate = CHECKED;
			}
			#ifdef DEBUG
				lprintf(lvl, "|-----> Event(%p)::sync_barrier() done waiting... state = %s\n", this, print_event_status(event_p->estate));
			#endif
		}
	}
	//release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::sync_barrier()\n", dev_id, id);
#endif
	return;
}

void Event::record_to_queue(CQueue_p Rr){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::record_to_queue() getting lock\n", dev_id, id);
#endif
	get_lock();
	if (Rr == NULL){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----> Event(%d)::record_to_queue(NULL)\n", dev_id, id);
#endif
		pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
		event_p->estate = CHECKED;
		status = CHECKED;
		release_lock();
		return;
	}
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::record_to_queue(Queue(dev_id=%d))\n", dev_id, id, Rr->dev_id);
#endif
	int prev_dev_id;
	hipGetDevice(&prev_dev_id);
	if (Rr->dev_id != prev_dev_id){
		CoCoPeLiaSelectDevice(Rr->dev_id);
#ifdef UDEBUG
		warning("Event(%d,dev_id = %d)::record_to_queue(%d): caller prev_dev_id=%d, changing to %d\n",
		id, dev_id, Rr->dev_id, prev_dev_id, Rr->dev_id);
#endif
	}
	if (status != UNRECORDED){
		;
#ifdef UDEBUG
		warning("Event(%d,dev_id = %d)::record_to_queue(%d): Recording %s event\n",
			id, dev_id, Rr->dev_id, print_event_status(status));
#endif
#ifdef ENABLE_LAZY_EVENTS
		if(Rr->dev_id != dev_id)
			error("(Lazy)Event(%d,dev_id = %d)::record_to_queue(%d): Recording %s event in iligal dev\n",
				id, dev_id, Rr->dev_id, print_event_status(status));
#endif
	}
#ifdef ENABLE_LAZY_EVENTS
	else if (status == UNRECORDED){
		if(dev_id > -1) /// TODO: This used to be an error, but with soft reset it was problematic...is it ok?
			;//warning("(Lazy)Event(%d,dev_id = %d)::record_to_queue(%d) - UNRECORDED event suspicious dev_id\n",
			//	id, dev_id, Rr->dev_id);
		dev_id = Rr->dev_id;
	}
#endif
	pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
	if(event_p->estate != UNRECORDED) {
		error("Event(%d,dev_id = %d)::record_to_queue(%d): Recording %s event\n",
			id, dev_id, Rr->dev_id, print_event_status(status));
		release_lock();
		return;
	}

	event_p->estate = RECORDED;
	status = RECORDED;
	if (Rr->dev_id != prev_dev_id){
		hipSetDevice(prev_dev_id);
	}
	release_lock();

	Rr->add_host_func((void*) &eventFunc, (void*) event_p, "eventFunc");
#ifdef DDEBUG
	lprintf(lvl, "Event(%p)::record_to_queue(Queue = %p)\n", this, Rr);
#endif

#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::record_to_queue(Queue(dev_id=%d))\n", dev_id, id, Rr->dev_id);
#endif
}

event_status Event::query_status(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::query_status()\n", dev_id, id);
#endif
	get_lock();
	enum event_status local_status = status;
	if (local_status != CHECKED){
#ifdef ENABLE_LAZY_EVENTS
		if (local_status == UNRECORDED){
			release_lock();
			return UNRECORDED;
		}
#endif
		pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
		
		if(status == RECORDED && event_p->estate == COMPLETE) status = COMPLETE;

		if(status != event_p->estate){
#ifdef UDDEBUG
			lprintf(lvl, "[dev_id=%3d] ------- Event(%d)::query_status() status = %s, event_p->estate = %s\n", dev_id, id, print_event_status(status), print_event_status(event_p->estate));
#endif
		}

		local_status = event_p->estate;
	}
	else {
		// local_status == CHECKED
		// update estate
		pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
		event_p->estate = CHECKED;
	}
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::query_status() = %s\n", dev_id, id, print_event_status(status));
#endif
	return local_status;
}

void Event::checked(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::checked()\n", dev_id, id);
#endif
	get_lock();
	if (status == COMPLETE) {
		status = CHECKED;
		pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
		event_p->estate = CHECKED;
	}
	else error("Event::checked(): error event was %s,  not COMPLETE()\n", print_event_status(status));
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::checked()\n", dev_id, id);
#endif
}

void Event::soft_reset(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::soft_reset()\n", dev_id, id);
#endif
	get_lock();
		// reset state
		pthread_event_p event_p = (pthread_event_p) event_backend_ptr;
		event_p->estate = UNRECORDED;
		status = UNRECORDED;
#ifdef ENABLE_LAZY_EVENTS
		if(dev_id >= -1){
			dev_id = dev_id - 42;
		}
#endif
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::soft_reset()\n", dev_id, id);
#endif
}

void Event::reset(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::reset() calls soft_reset()\n", dev_id, id);
#endif
#ifdef DDEBUG
	lprintf(lvl, "Event(%p)::reset started\n", this);
#endif

	sync_barrier();
	soft_reset();

#ifdef DDEBUG
	lprintf(lvl, "Event(%p)::reset done\n", this);
#endif

#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::reset()\n", dev_id, id);
#endif
}

/*****************************************************/
/// Event-based timer class functions

Event_timer::Event_timer(int dev_id) {
  Event_start = new Event(dev_id);
  Event_stop = new Event(dev_id);
  time_ms = 0;
}

void Event_timer::start_point(CQueue_p start_queue)
{
	Event_start->record_to_queue(start_queue);
}

void Event_timer::stop_point(CQueue_p stop_queue)
{
	Event_stop->record_to_queue(stop_queue);
}

double Event_timer::sync_get_time()
{
	if(Event_start->query_status() != UNRECORDED){
		Event_start->sync_barrier();
		if(Event_stop->query_status() != UNRECORDED) Event_stop->sync_barrier();
		else error("Event_timer::sync_get_time: Event_start is %s but Event_stop still UNRECORDED\n",
			print_event_status(Event_start->query_status()));
		
		pthread_event_p start_event = (pthread_event_p) Event_start->event_backend_ptr;
		pthread_event_p stop_event = (pthread_event_p) Event_stop->event_backend_ptr;

		time_ms = (double) std::chrono::duration_cast<std::chrono::milliseconds>(stop_event->completeTime - start_event->completeTime).count();
	}
	else time_ms = 0;
	return time_ms;
}

/*****************************************************/
