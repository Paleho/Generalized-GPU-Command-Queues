///
/// \author Poutas Sokratis (sokratispoutas@gmail.com)
///
/// \brief Wrapped host functions for firing cublas and cblas operations
///

#include <cblas.h>

#include "pthreads_backend_wrappers.hpp"

// void CoCoQueueLock(void* wrapped_lock){
// #ifdef ENABLE_MUTEX_LOCKING
//   (*(std::mutex*)wrapped_lock).lock();
// #else
//   while(__sync_lock_test_and_set ((&(*((int*)wrapped_lock))), 1));
// #endif
// #ifdef DEBUG
//   lprintf(6, "CoCoQueueLock(%p) ran succesfully.\n", wrapped_lock);
// #endif
// }

// void CoCoQueueUnlock(void* wrapped_lock){
// #ifdef ENABLE_MUTEX_LOCKING
// 	(*(std::mutex*)wrapped_lock).unlock();
// #else
//   //int* intptr = (int*) wrapped_lock;
//   //*intptr = 0;
//   __sync_lock_release((&(*((int*) wrapped_lock))));
// #endif

// #ifdef DEBUG
//   lprintf(6, "CoCoQueueUnlock(%p) ran succesfully.\n", wrapped_lock);
// #endif
// }

// void CoCoIncAsync(void* wrapped_ptr_int){
//   Ptr_atomic_int_p unwrapped = (Ptr_atomic_int_p) wrapped_ptr_int;
//   *(unwrapped->ato_int_ptr)++;
//   free(unwrapped);
// #ifdef DEBUG
//   lprintf(6, "CoCoIncAsync(%p, new_val=%d) ran succesfully.\n", unwrapped->ato_int_ptr, (*(unwrapped->ato_int_ptr)).load());
// #endif
// }

// void CoCoDecAsync(void* wrapped_ptr_int){
//   Ptr_atomic_int_p unwrapped = (Ptr_atomic_int_p) wrapped_ptr_int;
//   (*(unwrapped->ato_int_ptr))--;
//   free(unwrapped);
// #ifdef DEBUG
//   lprintf(6, "CoCoDecAsync(%p, new_val=%d) ran succesfully.\n", unwrapped->ato_int_ptr, (*(unwrapped->ato_int_ptr)).load());
// #endif
// }

// void CoCoSetInt(void* wrapped_ptr_and_val){
//   Ptr_and_int_p unwrapped = (Ptr_and_int_p) wrapped_ptr_and_val;
//   *(unwrapped->int_ptr) = unwrapped->val;
//   free(unwrapped);
// #ifdef DEBUG
//   lprintf(6, "CoCoSetVal(%p, %d) ran succesfully.\n", unwrapped->int_ptr, unwrapped->val);
// #endif
// }

// void CoCoSetPtr(void* wrapped_ptr_and_parent){
//   Ptr_and_parent_p unwrapped = (Ptr_and_parent_p) wrapped_ptr_and_parent;
//   void* prev_ptr = *(unwrapped->ptr_parent);
//   *(unwrapped->ptr_parent) = unwrapped->ptr_val;
//   free(unwrapped);
// #ifdef DEBUG
//   lprintf(6, "CoCoSetPtr(prev=%p, %p) ran succesfully.\n", prev_ptr, unwrapped->ptr_val);
// #endif
// }

// void CoCoSetTimerAsync(void* wrapped_timer_Ptr){
//   double* timer = (double*) wrapped_timer_Ptr;
//   *timer = csecond();
// #ifdef DEBUG
//   lprintf(6, "CoCoSetTimerAsync(%p) ran succesfully.\n", wrapped_timer_Ptr);
// #endif
// }

void cblas_wrap_daxpy(void* backend_data){
  axpy_backend_in<double>* ptr_ker_translate = (axpy_backend_in<double>*) backend_data;
  cblas_daxpy(ptr_ker_translate->N, ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->x, ptr_ker_translate->incx, (double*)
    *ptr_ker_translate->y, ptr_ker_translate->incy);
}

void cblas_wrap_saxpy(void* backend_data){
  axpy_backend_in<float>* ptr_ker_translate = (axpy_backend_in<float>*) backend_data;
  cblas_saxpy(ptr_ker_translate->N, ptr_ker_translate->alpha,
    (float*) *ptr_ker_translate->x, ptr_ker_translate->incx, (float*)
    *ptr_ker_translate->y, ptr_ker_translate->incy);
}

void cblas_wrap_ddot(void* backend_data){
  dot_backend_in<double>* ptr_ker_translate = (dot_backend_in<double>*) backend_data;
  *ptr_ker_translate->result = cblas_ddot(ptr_ker_translate->N, (double*) *ptr_ker_translate->x,
  ptr_ker_translate->incx, (double*) *ptr_ker_translate->y,
  ptr_ker_translate->incy);
}

void cblas_wrap_dgemm(void* backend_data){
  short lvl = 6;
  gemm_backend_in<double>* ptr_ker_translate = (gemm_backend_in<double>*) backend_data;
#ifdef DDEBUG
  if (ptr_ker_translate->dev_id != -1)
    warning("cblas_wrap_dgemm: Suspicious device %d instead of -1\n", ptr_ker_translate->dev_id);
#endif
#ifdef DDEBUG
  lprintf(lvl, "cblas_wrap_dgemm: cblas_dgemm(dev_id = %d, TransA = %c, TransB = %c,\
    M = %d, N = %d, K = %d, alpha = %lf, A = %p, lda = %d, \n\
    B = %p, ldb = %d, beta = %lf, C = %p, ldC = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA, ptr_ker_translate->TransB,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (VALUE_TYPE*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (VALUE_TYPE*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (VALUE_TYPE*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
#endif
  cblas_dgemm(CblasColMajor,
    OpCharToCblas(ptr_ker_translate->TransA), OpCharToCblas(ptr_ker_translate->TransB),
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (double*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (double*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
}

void cblas_wrap_dgemv(void* backend_data){
  short lvl = 6;
  gemv_backend_in<double>* ptr_ker_translate = (gemv_backend_in<double>*) backend_data;
#ifdef DDEBUG
  if (ptr_ker_translate->dev_id != -1)
    warning("cblas_wrap_dgemv: Suspicious device %d instead of -1\n", ptr_ker_translate->dev_id);
#endif
#ifdef DDEBUG
  lprintf(lvl, "cblas_wrap_dgemv: cblas_dgemv(dev_id = %d, TransA = %c\
    M = %d, N = %d,alpha = %lf, A = %p, lda = %d, \n\
    beta = %lf, x = %p, incx = %d, y = %p, incy = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (double*) *ptr_ker_translate->x, ptr_ker_translate->incx,
    ptr_ker_translate->beta, (double*) *ptr_ker_translate->y, ptr_ker_translate->incy);
#endif
  cblas_dgemv(CblasColMajor,
    OpCharToCblas(ptr_ker_translate->TransA),
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (double*) *ptr_ker_translate->x, ptr_ker_translate->incx,
    ptr_ker_translate->beta, (double*) *ptr_ker_translate->y, ptr_ker_translate->incy);
}

void cblas_wrap_sgemm(void* backend_data){
  short lvl = 6;
  gemm_backend_in<float>* ptr_ker_translate = (gemm_backend_in<float>*) backend_data;
#ifdef DDEBUG
  if (ptr_ker_translate->dev_id != -1)
    warning("cblas_wrap_sgemm: Suspicious device %d instead of -1\n", ptr_ker_translate->dev_id);
#endif
#ifdef DDEBUG
  lprintf(lvl, "cblas_wrap_sgemm: cblas_dgemm(dev_id = %d, TransA = %c, TransB = %c,\
    M = %d, N = %d, K = %d, alpha = %lf, A = %p, lda = %d, \n\
    B = %p, ldb = %d, beta = %lf, C = %p, ldC = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA, ptr_ker_translate->TransB,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (float*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (float*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (float*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
#endif
  cblas_sgemm(CblasColMajor,
    OpCharToCblas(ptr_ker_translate->TransA), OpCharToCblas(ptr_ker_translate->TransB),
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (float*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (float*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (float*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
}

void cublas_wrap_daxpy(void* wider_backend_data){
  wider_backend_in_p wider_data_p = (wider_backend_in_p) wider_backend_data;
  
  axpy_backend_in<double>* ptr_ker_translate = (axpy_backend_in<double>*) wider_data_p->backend_data;
  CoCoPeLiaSelectDevice(ptr_ker_translate->dev_id);

  queue_data_p queue_backend_data = wider_data_p->q_data;

  get_lock_q(&queue_backend_data->queueLock);
    hipblasHandle_t temp_handle = *(queue_backend_data->handle_p);

    // Get stream and increase stream index
    int current_stream_ctr = queue_backend_data->stream_ctr;
    queue_backend_data->stream_ctr = (current_stream_ctr + 1) % STREAM_POOL_SZ;
	release_lock_q(&queue_backend_data->queueLock);

  // Set stream
  massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(temp_handle, queue_backend_data->stream_pool[current_stream_ctr]),
		"cublas_wrap_daxpy: hipblasSetStream failed\n", ptr_ker_translate->dev_id);


  massert(HIPBLAS_STATUS_SUCCESS == hipblasDaxpy(temp_handle,
    ptr_ker_translate->N, (double*) &ptr_ker_translate->alpha, (double*) *ptr_ker_translate->x,
    ptr_ker_translate->incx, (double*) *ptr_ker_translate->y, ptr_ker_translate->incy),
    "cublas_wrap_daxpy failed\n");

  hipError_t err = hipStreamSynchronize(queue_backend_data->stream_pool[current_stream_ctr]);
  massert(hipSuccess == err, "cublas_wrap_daxpy: stream sync failed, current_stream_ctr = %d, queue_backend_data = %x, queue_backend_data->stream_pool[current_stream_ctr] = %x, hipError_t = %s\n", current_stream_ctr, queue_backend_data, queue_backend_data->stream_pool[current_stream_ctr], hipGetErrorString(err));
}

void cublas_wrap_ddot(void* wider_backend_data){
  wider_backend_in_p wider_data_p = (wider_backend_in_p) wider_backend_data;

  dot_backend_in<double>* ptr_ker_translate = (dot_backend_in<double>*) wider_data_p->backend_data;
  CoCoPeLiaSelectDevice(ptr_ker_translate->dev_id);

  queue_data_p queue_backend_data = wider_data_p->q_data;

  get_lock_q(&queue_backend_data->queueLock);
    hipblasHandle_t temp_handle = *(queue_backend_data->handle_p);

    // Get stream and increase stream index
    int current_stream_ctr = queue_backend_data->stream_ctr;
    queue_backend_data->stream_ctr = (current_stream_ctr + 1) % STREAM_POOL_SZ;
	release_lock_q(&queue_backend_data->queueLock);

  // Set stream
  massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(temp_handle, queue_backend_data->stream_pool[current_stream_ctr]),
		"cublas_wrap_ddot: hipblasSetStream failed\n", ptr_ker_translate->dev_id);

  massert(HIPBLAS_STATUS_SUCCESS == hipblasDdot(temp_handle, ptr_ker_translate->N,
      (double*) *ptr_ker_translate->x, ptr_ker_translate->incx, (double*) *ptr_ker_translate->y,
      ptr_ker_translate->incy, (double*)ptr_ker_translate->result),
    "cublas_wrap_ddot failed\n");

  hipError_t err = hipStreamSynchronize(queue_backend_data->stream_pool[current_stream_ctr]);
  massert(hipSuccess == err, "cublas_wrap_ddot: stream sync failed, current_stream_ctr = %d, queue_backend_data = %x, queue_backend_data->stream_pool[current_stream_ctr] = %x, hipError_t = %s\n", current_stream_ctr, queue_backend_data, queue_backend_data->stream_pool[current_stream_ctr], hipGetErrorString(err));
}

void cublas_wrap_dgemm(void* wider_backend_data){
  short lvl = 6;
  wider_backend_in_p wider_data_p = (wider_backend_in_p) wider_backend_data;

  gemm_backend_in<double>* ptr_ker_translate = (gemm_backend_in<double>*) wider_data_p->backend_data;
#ifdef DDEBUG
  int cur_dev_id = CoCoPeLiaGetDevice();
  if (ptr_ker_translate->dev_id != cur_dev_id)
    warning("cublas_wrap_dgemm: Changing device %d -> %d\n", cur_dev_id, ptr_ker_translate->dev_id);
#endif
  CoCoPeLiaSelectDevice(ptr_ker_translate->dev_id);
#ifdef DDEBUG
  lprintf(lvl, "cublas_wrap_dgemm: hipblasDgemm(dev_id = %d, TransA = %c, TransB = %c,\
    M = %d, N = %d, K = %d, alpha = %lf, A = %p, lda = %d, \n\
    B = %p, ldb = %d, beta = %lf, C = %p, ldC = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA, ptr_ker_translate->TransB,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (double*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (double*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
#endif

  queue_data_p queue_backend_data = wider_data_p->q_data;

  get_lock_q(&queue_backend_data->queueLock);
    hipblasHandle_t temp_handle = *(queue_backend_data->handle_p);

    // Get stream and increase stream index
    int current_stream_ctr = queue_backend_data->stream_ctr;
    queue_backend_data->stream_ctr = (current_stream_ctr + 1) % STREAM_POOL_SZ;
	release_lock_q(&queue_backend_data->queueLock);

  // Set stream
  massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(temp_handle, queue_backend_data->stream_pool[current_stream_ctr]),
		"cublas_wrap_dgemm: hipblasSetStream failed\n", ptr_ker_translate->dev_id);

  massert(HIPBLAS_STATUS_SUCCESS == hipblasDgemm(temp_handle,
    OpCharToCublas(ptr_ker_translate->TransA), OpCharToCublas(ptr_ker_translate->TransB),
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, &ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (double*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    &ptr_ker_translate->beta, (double*) *ptr_ker_translate->C, ptr_ker_translate->ldC),
    "cublas_wrap_dgemm: hipblasDgemm failed\n");

  hipError_t err = hipStreamSynchronize(queue_backend_data->stream_pool[current_stream_ctr]);
  massert(hipSuccess == err, "cublas_wrap_dgemm: stream sync failed, current_stream_ctr = %d, queue_backend_data = %x, queue_backend_data->stream_pool[current_stream_ctr] = %x, hipError_t = %s\n", current_stream_ctr, queue_backend_data, queue_backend_data->stream_pool[current_stream_ctr], hipGetErrorString(err));
}

void cublas_wrap_sgemm(void* wider_backend_data){
  short lvl = 6;
  wider_backend_in_p wider_data_p = (wider_backend_in_p) wider_backend_data;

  gemm_backend_in<float>* ptr_ker_translate = (gemm_backend_in<float>*) wider_data_p->backend_data;
#ifdef DDEBUG
  int cur_dev_id = CoCoPeLiaGetDevice();
  if (ptr_ker_translate->dev_id != cur_dev_id)
    warning("cublas_wrap_sgemm: Changing device %d -> %d\n", cur_dev_id, ptr_ker_translate->dev_id);
#endif
  CoCoPeLiaSelectDevice(ptr_ker_translate->dev_id);
#ifdef DDEBUG
  lprintf(lvl, "cublas_wrap_sgemm: hipblasDgemm(dev_id = %d, TransA = %c, TransB = %c,\
    M = %d, N = %d, K = %d, alpha = %lf, A = %p, lda = %d, \n\
    B = %p, ldb = %d, beta = %lf, C = %p, ldC = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA, ptr_ker_translate->TransB,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (float*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (float*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (float*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
#endif

  queue_data_p queue_backend_data = wider_data_p->q_data;

  get_lock_q(&queue_backend_data->queueLock);
    hipblasHandle_t temp_handle = *(queue_backend_data->handle_p);

    // Get stream and increase stream index
    int current_stream_ctr = queue_backend_data->stream_ctr;
    queue_backend_data->stream_ctr = (current_stream_ctr + 1) % STREAM_POOL_SZ;
	release_lock_q(&queue_backend_data->queueLock);

  // Set stream
  massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(temp_handle, queue_backend_data->stream_pool[current_stream_ctr]),
		"cublas_wrap_sgemm: hipblasSetStream failed\n", ptr_ker_translate->dev_id);

  massert(HIPBLAS_STATUS_SUCCESS == hipblasSgemm(temp_handle,
    OpCharToCublas(ptr_ker_translate->TransA), OpCharToCublas(ptr_ker_translate->TransB),
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, &ptr_ker_translate->alpha,
    (float*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (float*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    &ptr_ker_translate->beta, (float*) *ptr_ker_translate->C, ptr_ker_translate->ldC),
    "cublas_wrap_sgemm: hipblasSgemm failed\n");

  hipError_t err = hipStreamSynchronize(queue_backend_data->stream_pool[current_stream_ctr]);
  massert(hipSuccess == err, "cublas_wrap_sgemm: stream sync failed, current_stream_ctr = %d, queue_backend_data = %x, queue_backend_data->stream_pool[current_stream_ctr] = %x, hipError_t = %s\n", current_stream_ctr, queue_backend_data, queue_backend_data->stream_pool[current_stream_ctr], hipGetErrorString(err));
}

void cublas_wrap_dgemv(void* wider_backend_data){
  short lvl = 6;
  wider_backend_in_p wider_data_p = (wider_backend_in_p) wider_backend_data;

  gemv_backend_in<double>* ptr_ker_translate = (gemv_backend_in<double>*) wider_data_p->backend_data;
#ifdef DDEBUG
  int cur_dev_id = CoCoPeLiaGetDevice();
  if (ptr_ker_translate->dev_id != cur_dev_id)
    warning("cublas_wrap_dgemv: Changing device %d -> %d\n", cur_dev_id, ptr_ker_translate->dev_id);
#endif
#ifdef DDEBUG
  lprintf(lvl, "cublas_wrap_dgemv: cblas_dgemv(dev_id = %d, TransA = %c\
    M = %d, N = %d,alpha = %lf, A = %p, lda = %d, \n\
    beta = %lf, x = %p, incx = %d, y = %p, incy = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (double*) *ptr_ker_translate->x, ptr_ker_translate->incx,
    ptr_ker_translate->beta, (double*) *ptr_ker_translate->y, ptr_ker_translate->incy);
#endif

  queue_data_p queue_backend_data = wider_data_p->q_data;

  get_lock_q(&queue_backend_data->queueLock);
    hipblasHandle_t temp_handle = *(queue_backend_data->handle_p);

    // Get stream and increase stream index
    int current_stream_ctr = queue_backend_data->stream_ctr;
    queue_backend_data->stream_ctr = (current_stream_ctr + 1) % STREAM_POOL_SZ;
	release_lock_q(&queue_backend_data->queueLock);

  // Set stream
  massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(temp_handle, queue_backend_data->stream_pool[current_stream_ctr]),
		"cublas_wrap_dgemv: hipblasSetStream failed\n", ptr_ker_translate->dev_id);

  massert(HIPBLAS_STATUS_SUCCESS == hipblasDgemv(temp_handle, OpCharToCublas(ptr_ker_translate->TransA),
    ptr_ker_translate->M, ptr_ker_translate->N, &ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (double*) *ptr_ker_translate->x, ptr_ker_translate->incx,
    &ptr_ker_translate->beta, (double*) *ptr_ker_translate->y, ptr_ker_translate->incy),
    "cublas_wrap_dgemv: hipblasDgemv failed\n");

  hipError_t err = hipStreamSynchronize(queue_backend_data->stream_pool[current_stream_ctr]);
  massert(hipSuccess == err, "cublas_wrap_dgemv: stream sync failed, current_stream_ctr = %d, queue_backend_data = %x, queue_backend_data->stream_pool[current_stream_ctr] = %x, hipError_t = %s\n", current_stream_ctr, queue_backend_data, queue_backend_data->stream_pool[current_stream_ctr], hipGetErrorString(err));
}
