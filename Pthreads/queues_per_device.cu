#include <hip/hip_runtime.h>
#include "queues_per_device.hpp"

std::vector<CommandQueue *> * queuesPerDevice;
bool queuesPerDeviceInitialized = false;

void InitializeQueuesPerDevice(){
    if(queuesPerDeviceInitialized){
        warning("InitializeQueuesPerDevice(): already initialized\n");
        return;
    }

    // Get number of devices
    int dev_count;
    hipError_t err = hipGetDeviceCount(&dev_count);
    massert(hipSuccess == err, "InitializeQueuesPerDevice(): hipGetDeviceCount() failed - %s\n", hipGetErrorString(err));

    queuesPerDevice = new std::vector<CommandQueue *>[dev_count]();
    if(!queuesPerDevice) error("InitializeQueuesPerDevice(): malloc failed\n");

    queuesPerDeviceInitialized = true;
}

void UninitializeQueuesPerDevice(){
    if(!queuesPerDeviceInitialized){
        error("UninitializeQueuesPerDevice(%): cannot destroy, structure is not initialized\n");
    }

    delete [] queuesPerDevice;

    queuesPerDeviceInitialized = false;
}

void AssignQueueToDevice(CommandQueue * queue, int dev){
#ifdef DEBUG
	lprintf(1, "AssignQueueToDevice(%p, %d)\n", queue, dev);
#endif
    if(!queuesPerDeviceInitialized){
        error("AssignQueueToDevice(%p, %d): cannot assign, structure is not initialized (call InitializeQueuesPerDevice() first)\n", queue, dev);
    }
    // "Host" device loc id used by CoCoPeLia is 0. See CoCoPeLiaSelectDevice
    int inner_dev_id = (dev == -1) ? 0: dev;

    queuesPerDevice[inner_dev_id].push_back(queue);
}

void UnassignQueueFromDevice(CommandQueue * queue, int dev){
    if(!queuesPerDeviceInitialized){
        error("UnassignQueueFromDevice(%p, %d): cannot unassign, structure is not initialized\n", queue, dev);
    }
    // "Host" device loc id used by CoCoPeLia is 0. See CoCoPeLiaSelectDevice
    int inner_dev_id = (dev == -1) ? 0: dev;

	for(int i = 0; i < queuesPerDevice[inner_dev_id].size(); i++){
		if(queuesPerDevice[inner_dev_id][i] == queue) queuesPerDevice[inner_dev_id].erase(queuesPerDevice[inner_dev_id].begin()+i);
	}
}

void DeviceSynchronize(){
    if(!queuesPerDeviceInitialized){
        // error("DeviceSynchronize(): cannot synchronize queues, structure is not initialized (call InitializeQueuesPerDevice() first)\n");
        return;
    }

    int dev = -1;
    hipError_t err = hipGetDevice(&dev);
    massert(hipSuccess == err,"DeviceSynchronize(): hipGetDevice failed - %s\n", hipGetErrorString(err));

    for(int i = 0; i < queuesPerDevice[dev].size(); i++){
        queuesPerDevice[dev][i]->sync_barrier();
    }
}
